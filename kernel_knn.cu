#include "hip/hip_runtime.h"
//The distKernel computes the difference squared between two points. Requires size number of threads
// input:	2d dimentional data, input x (read only data path)
// output:	distance for each sample/row of 2d data 

// version 1: earch row/thread cacludate the difference
//				inX  is in the read-only data path
__global__ void distKernel(float *inX, float *dataSet, int row, int col, float *distance)
{
	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	if (gid < rows)
	{
		float dist = 0.f;

		for(int i=0; i<col; i++)
		{
			float f = (inX[i] - dataSet[gid * col + i]) 
			dist += f * f;
		}
		distance[gid] = sqrt(dist);
	}
}
